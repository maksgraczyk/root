/*
 * Project: RooFit
 * Author:
 *   Jonas Rembser, CERN 2023
 *
 * Copyright (c) 2023, CERN
 *
 * Redistribution and use in source and binary forms,
 * with or without modification, are permitted according to the terms
 * listed in LICENSE (http://roofit.sourceforge.net/license.txt)
 */

#include <RooFit/Detail/CudaInterface.h>

#include <stdexcept>
#include <sstream>
#include <string>

#define ERRCHECK(err) __checkCudaErrors((err), __func__, __FILE__, __LINE__)
inline static void __checkCudaErrors(hipError_t error, std::string func, std::string file, int line)
{
   if (error != hipSuccess) {
      std::stringstream errMsg;
      errMsg << func << "(), " << file + ":" << std::to_string(line) << " : " << hipGetErrorString(error);
      throw std::runtime_error(errMsg.str());
   }
}

namespace RooFit {
namespace Detail {
namespace CudaInterface {

DeviceMemory::DeviceMemory(std::size_t n, std::size_t typeSize) : _size{n}
{
   void *ret;
   ERRCHECK(::hipMalloc(&ret, n * typeSize));
   _data.reset(ret);
}
PinnedHostMemory::PinnedHostMemory(std::size_t n, std::size_t typeSize) : _size{n}
{
   void *ret;
   ERRCHECK(::hipHostMalloc(&ret, n * typeSize));
   _data.reset(ret);
}

template <>
void Deleter<DeviceMemory>::operator()(void *ptr)
{
   ERRCHECK(::hipFree(ptr));
}
template <>
void Deleter<PinnedHostMemory>::operator()(void *ptr)
{
   ERRCHECK(::hipHostFree(ptr));
}

/**
 * Creates a new CUDA event.
 *
 * @param[in] forTiming       Set to true if the event is intended for timing purposes.
 *                            If `false`, the `hipEventDisableTiming` is passed to CUDA.
 * @return                    CudaEvent object representing the new event.
 */
CudaEvent newCudaEvent(bool forTiming)
{
   CudaEvent ret;
   ret.reset(new hipEvent_t);
   ERRCHECK(hipEventCreateWithFlags(ret.get(), forTiming ? 0 : hipEventDisableTiming));
   return ret;
}

/**
 * Destroys a CUDA event.
 *
 * @param[in] event           CudaEvent object representing the event to be destroyed.
 */
void deleteCudaEvent(CudaEvent event)
{
   ERRCHECK(hipEventDestroy(*event.get()));
   delete event.get();
   event.reset(nullptr);
}

/**
 * Records a CUDA event.
 *
 * @param[in] event           CudaEvent object representing the event to be recorded.
 * @param[in] stream          CudaStream in which to record the event.
 */
void hipEventRecord(CudaEvent event, CudaStream stream)
{
   ERRCHECK(::hipEventRecord(*event.get(), *stream.get()));
}

/**
 * Creates a new CUDA stream.
 *
 * @return                    CudaStream object representing the new stream.
 */
CudaStream newCudaStream()
{
   CudaStream ret;
   ret.reset(new hipStream_t);
   ERRCHECK(hipStreamCreate(ret.get()));
   return ret;
}

/**
 * Destroys a CUDA stream.
 *
 * @param[in] stream          CudaStream object representing the stream to be destroyed.
 */
void deleteCudaStream(CudaStream stream)
{
   ERRCHECK(hipStreamDestroy(*stream.get()));
   delete stream.get();
   stream.reset(nullptr);
}

/**
 * Checks if a CUDA stream is currently active.
 *
 * @param[in] stream          CudaStream object representing the stream to be checked.
 * @return                    True if the stream is active, false otherwise.
 */
bool streamIsActive(CudaStream stream)
{
   hipError_t err = hipStreamQuery(*stream.get());
   if (err == hipErrorNotReady)
      return true;
   else if (err == hipSuccess)
      return false;
   ERRCHECK(err);
   return false;
}

/**
 * Makes a CUDA stream wait for a CUDA event.
 *
 * @param[in] stream          CudaStream object representing the stream to wait on.
 * @param[in] event           CudaEvent object representing the event to wait for.
 */
void hipStreamWaitEvent(CudaStream stream, CudaEvent event)
{
   ERRCHECK(::hipStreamWaitEvent(*stream.get(), *event.get(), 0));
}

/**
 * Calculates the elapsed time between two CUDA events.
 *
 * @param[in] begin           CudaEvent representing the start event.
 * @param[in] end             CudaEvent representing the end event.
 * @return                    Elapsed time in milliseconds.
 */
float hipEventElapsedTime(CudaEvent begin, CudaEvent end)
{
   float ret;
   ERRCHECK(::hipEventElapsedTime(&ret, *begin.get(), *end.get()));
   return ret;
}

/// \internal
void copyHostToDeviceImpl(const void *src, void *dest, size_t nBytes, CudaStream stream)
{
   if (stream.get())
      ERRCHECK(hipMemcpyAsync(dest, src, nBytes, hipMemcpyHostToDevice, *stream.get()));
   else
      ERRCHECK(hipMemcpy(dest, src, nBytes, hipMemcpyHostToDevice));
}

/// \internal
void copyDeviceToHostImpl(const void *src, void *dest, size_t nBytes, CudaStream stream)
{
   if (stream.get())
      ERRCHECK(hipMemcpyAsync(dest, src, nBytes, hipMemcpyDeviceToHost, *stream.get()));
   else
      ERRCHECK(hipMemcpy(dest, src, nBytes, hipMemcpyDeviceToHost));
}

} // namespace CudaInterface
} // namespace Detail
} // namespace RooFit
